#include "hip/hip_runtime.h"
%%cuda
//Colab link:https://colab.research.google.com/drive/17VkDTCBgf8RmIdYF-2xNw7xzXiI44rEp?usp=sharing
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>
#define size 1000000
using namespace std;

//DEVICE
__global__ void kernelVector_x_constant(float* arrayA, float* arrayB, float* sum_arr, int n) {
    // Get the index of the thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // If the thread index is valid
    if (idx < n) {
        // Add the elements from both arrays
        sum_arr[idx] = arrayA[idx] + arrayB[idx];
    }
}

// Function to check for CUDA errors
inline hipError_t checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "CUDA Runtime Error: " << hipGetErrorString(result) << endl;
        exit(-1);
    }
    return result;
}


//HOST
int main() {
    // Allocate memory on the host
    float* arrayA = new float[size];
    float* arrayB = new float[size];
    float* sum_arr = new float[size];
    float* arrayA_DEVICE = NULL;
    float* arrayB_DEVICE = NULL;
    float* sum_arr_DEVICE = NULL;

    // Initialize the arrays on the host
    for (int index = 0; index < size; index++) {
        arrayA[index] = index;
        arrayB[index] = index;
    }

    // Measure host sum time using ctime
    clock_t start_host = clock();
    // Sum on host
    for (int index = 0; index < size; index++) {
        sum_arr[index] = arrayA[index] + arrayB[index];
    }
    clock_t end_host = clock();
    double duration_host = double(end_host - start_host) / CLOCKS_PER_SEC;

    // Print the host first 100 results
    cout << "Host results: " << endl;
    for (int index = 0; index < 100; index++) {
        cout << sum_arr[index] << endl;
    }

    // Allocate memory on the device (same amount of bytes as on the host)
    checkCuda(hipMalloc((void**)&arrayA_DEVICE, size * sizeof(float)));
    checkCuda(hipMalloc((void**)&arrayB_DEVICE, size * sizeof(float)));
    checkCuda(hipMalloc((void**)&sum_arr_DEVICE, size * sizeof(float)));

    // Copy the memory block from the host to the device
    checkCuda(hipMemcpy(arrayA_DEVICE, arrayA, size * sizeof(float), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(arrayB_DEVICE, arrayB, size * sizeof(float), hipMemcpyHostToDevice));

    // Initilize hipEvent_t variables to measure device sum tume
    hipEvent_t start_device, stop_device;
    checkCuda(hipEventCreate(&start_device));
    checkCuda(hipEventCreate(&stop_device));

    checkCuda(hipEventRecord(start_device));
    // Execute CUDA kernel with 512 threads and 1000000/12 blocks
    kernelVector_x_constant<<<ceil(size/512), 512>>>(arrayA_DEVICE, arrayB_DEVICE, sum_arr_DEVICE, size);
    checkCuda(hipEventRecord(stop_device));
    checkCuda(hipEventSynchronize(stop_device));
    
    float duration_device = 0;
    checkCuda(hipEventElapsedTime(&duration_device, start_device, stop_device));

    // Copy the processed data back to the host
    checkCuda(hipMemcpy(sum_arr, sum_arr_DEVICE, size * sizeof(float), hipMemcpyDeviceToHost));

    // Print the device first 100 results
    cout << "Device results: " << endl;
    for (int index = 0; index < 100; index++) {
        cout << sum_arr[index] << endl;
    }
    // Print execution times
    cout << "Host computation time: " << duration_host << " seconds" << endl;
    cout << "Device computation time: " << duration_device / 1000.0 << " seconds" << endl;

    // Free memory on the device
    hipFree(sum_arr_DEVICE);
    hipFree(arrayA_DEVICE);
    hipFree(arrayB_DEVICE);
    // Free memory on the host
    delete[] arrayA;
    delete[] arrayB;
    delete[] sum_arr;
    // Destroy CUDA events
    hipEventDestroy(start_device);
    hipEventDestroy(stop_device);

    return 0;
}
